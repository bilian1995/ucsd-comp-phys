#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
using std::cout;
using std::endl;

#include<fstream>
using std::ofstream;

#include <algorithm>
using std::fill;


/***** ERROR CHECKING MACRO *****/
hipError_t _TempErrorCode;
#define CHECK_CUDA_ERROR() _TempErrorCode = hipGetLastError(); if(_TempErrorCode) fprintf(stderr,"!!CUDA ERROR in %s at line %d : %s\n",__FILE__,__LINE__,hipGetErrorString(_TempErrorCode));



/***** CUSTOM COMMAND LINE ARGUMENT PARSING *****/
//list of global variables (with default values)
int NumberOfArgs = 1; //how many constants are listed below

// you can add your own global variables to be parsed here
// (I start with underline to distiguish that it is a global variable):
int   _ArraySize = 1024;
float _IncrementValue = 1.0f;
char _OutputFile[] = "output.txt";


//this will display the global variable values before program starts running
void displayGlobals(void){
	cout<<"Setting ArraysSize to " << _ArraySize << endl;
	cout<<"Setting IncrementValue to " << _IncrementValue << endl;
}


//this parses the command line arguments
void parseArguments(int arg_count, char* args[]){
	
	//the first argument is always the program
	cout << "Running (" << args[0] << ")" << endl;
	
	if(arg_count > NumberOfArgs){
		// add your string to whatever parsing here
		_ArraySize = atoi(args[1]);
		_IncrementValue = atof(args[2]);
		//for strings just copy the pointer? (address):
		//OutputFile = args[3];
		displayGlobals();
		
	}else{
		//output usage
		cout << "Usage: "<< args[0] << " <ArraySize> <IncrementValue> " << endl;// <OutputFile>" << endl;
		//show default values
		displayGlobals();
	}

}

/***** A DEVICE FUNCTION *****/ 
__device__ float AddNum(float a, float b){
	return a + b;
}

/***** CUDA KERNEL ******/

/** this function increments the inArray by increment for all indicies less than MaxIndex **/
__global__ void incrementKernel(float* outArray,float* inArray, int MaxIndex, float increment){
	
	//the objects (gridDim,blockIdx,blockDim,threadIdx) are already defined:
	int threadIndex = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(threadIndex < MaxIndex){ //keep it safe
		outArray[threadIndex] = AddNum(inArray[threadIndex], increment);
	}
	
}


/***** MAIN *****/
int main(int argc, char* argv[]){

	parseArguments(argc, argv); //this will set the global variables

	//Device array pointers
	float* inArray_dev; //set to zero to avoid compile warnings
	float* outArray_dev;
	
	//Host array pointers
	float* inArray_host;
	float* outArray_host;
	
	//initialize arrays on host (using c++)
	inArray_host = new float[_ArraySize]; //equiv. to (float*)malloc(sizeof(float)*ARRAY_SIZE);
	outArray_host = new float[_ArraySize];
	//fill
	fill(inArray_host, inArray_host+_ArraySize, 1.0f); //fill with ones
	fill(outArray_host, outArray_host+_ArraySize, 0.0f); //fill with zeros
	
	//initialize arrays on device (GPU)
	hipMalloc((void**)&inArray_dev, sizeof(float)*_ArraySize);
	CHECK_CUDA_ERROR();
	hipMalloc((void**)&outArray_dev, sizeof(float)*_ArraySize);
	CHECK_CUDA_ERROR();
	
	//fill
	hipMemset(inArray_dev, sizeof(float)*_ArraySize, 0); //set input array to zero (must be a byte value)
	CHECK_CUDA_ERROR();
	hipMemset(outArray_dev, sizeof(float)*_ArraySize, 0); //set output array to zero (must be a byte value)
	CHECK_CUDA_ERROR();
	
	//copy input array to device
	//hipMemcpy(DestinationPointer, SourcePointer, NumberOfBytes, hipMemcpy[Host|Device]To[Host|Device]);
	hipMemcpy(inArray_dev, inArray_host, sizeof(float)*_ArraySize, hipMemcpyHostToDevice);
	
	//__LAUNCH KERNEL__
	//in general this geometry can be 3D, but for now we are just indexing a linear array
	int threadsPerBlock = 512; //this is typically the max for most GPUs except Fermi
	int blockCount;
	
	//special case for small array size:
	if(_ArraySize <= threadsPerBlock){
		blockCount = 1;
	}else{
		blockCount = _ArraySize/threadsPerBlock + 1; //max block size
	}
	
	incrementKernel <<< blockCount,threadsPerBlock >>> (outArray_dev, inArray_dev, _ArraySize, _IncrementValue);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR();

	// copy back results
	hipMemcpy(outArray_host, outArray_dev, sizeof(float)*_ArraySize, hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR();

	ofstream outFileStream;
	outFileStream.open(_OutputFile);
	
	//print output
	for(int i = 0; i<_ArraySize; i++){
		outFileStream << outArray_host[i] << endl;
	}

	cout << "Data saved in file " << _OutputFile << endl;

	//cleanup
	outFileStream.close();
	delete inArray_host; //like free()
	delete outArray_host;
	hipFree(inArray_dev);
	hipFree(outArray_dev);
	
}
